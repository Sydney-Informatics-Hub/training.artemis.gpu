
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_world_kernel()
{
	printf("Hello GPU World!\n");
}

int main()
{
	hello_world_kernel <<<1,8>>>();
	hipDeviceReset();
}

